#include "hip/hip_runtime.h"
#include "../include/utils.h"
#include <hip/hip_runtime.h>

#define NUM_RUNS 10

#define CUDA_CHECK(func)                                                     	   \
	do {                                                                           \
		hipError_t status = (func);                                               \
		if (status != hipSuccess) {                                               \
			printf("CUDA API failed at line %d with error: %s (%d)\n", __LINE__,   \
				hipGetErrorString(status), status);                               \
			exit(EXIT_FAILURE);                                                    \
		}                                                                          \
	} while (0)

#define CHECK(name) \
	float *d_Aref_ ## name, *d_Bref_ ## name, *d_Cref_ ## name; \
	std::cerr << "checking " << #name << std::endl; \
	CUDA_CHECK(hipMalloc(&d_Aref_ ## name, Ref::M * Ref::K * sizeof(float))); \
	CUDA_CHECK(hipMalloc(&d_Bref_ ## name, Ref::K * Ref::N * sizeof(float))); \
	CUDA_CHECK(hipMalloc(&d_Cref_ ## name, Ref::M * Ref::N * sizeof(float))); \
	CUDA_CHECK(hipMemcpy(d_Aref_ ## name, ref.A, Ref::M * Ref::K * sizeof(float), hipMemcpyHostToDevice)); \
	CUDA_CHECK(hipMemcpy(d_Bref_ ## name, ref.B, Ref::K * Ref::N * sizeof(float), hipMemcpyHostToDevice)); \
	float* d_Cref_INI_ ## name = new float[M * N](); \
	for (int i = 0; i < Ref::M; i++) { \
		for (int j = 0; j < Ref::N; j++) { \
			d_Cref_INI_ ## name[i * Ref::N + j] = 0; \
		} \
	} \
	CUDA_CHECK(hipMemcpy(d_Cref_ ## name, d_Cref_INI_ ## name, Ref::M * Ref::N * sizeof(float), hipMemcpyHostToDevice)); \
	name(d_Aref_ ## name, d_Bref_ ## name, d_Cref_ ## name, Ref::M, Ref::N, Ref::K); \
	hipError_t err_c_ ## name = hipGetLastError(); \
	if (err_c_ ## name != hipSuccess) { \
		std::cerr << "CUDA Error: " << hipGetErrorString(err_c_ ## name) << std::endl; \
	} \
	CUDA_CHECK(hipMemcpy(refC, d_Cref_ ## name, Ref::M * Ref::N * sizeof(float), hipMemcpyDeviceToHost)); \
	if (!ref.checkRef(refC)){ \
		std::cerr << "check ref failed!" << std::endl; \
	};

#define TIME(name) \
	float *d_A_ ## name, *d_B_ ## name, *d_C_ ## name; \
	CUDA_CHECK(hipMalloc(&d_A_ ## name, M * K * sizeof(float))); \
	CUDA_CHECK(hipMalloc(&d_B_ ## name, K * N * sizeof(float))); \
	CUDA_CHECK(hipMalloc(&d_C_ ## name, M * N * sizeof(float))); \
	CUDA_CHECK(hipMemcpy(d_A_ ## name, A, M * K * sizeof(float), hipMemcpyHostToDevice)); \
	CUDA_CHECK(hipMemcpy(d_B_ ## name, B, K * N * sizeof(float), hipMemcpyHostToDevice)); \
	hipEvent_t start_ ## name, end_ ## name; \
	hipEventCreate(&start_ ## name); \
	hipEventCreate(&end_ ## name); \
	float* d_C_INI_ ## name = new float[M * N](); \
	for (int i = 0; i < M; i++) { \
		for (int j = 0; j < N; j++) { \
			d_C_INI_ ## name[i * N + j] = 0; \
		} \
	} \
	for (int i = 0; i < 2; i++) \
	{ \
		CUDA_CHECK(hipMemcpy(d_C_ ## name, d_C_INI_ ## name, M * N * sizeof(float), hipMemcpyHostToDevice)); \
		name(d_A_ ## name, d_B_ ## name, d_C_ ## name, M, N, K); \
	} \
	hipError_t err_t_ ## name = hipGetLastError(); \
	if (err_t_ ## name != hipSuccess) { \
		std::cerr << "CUDA Error: " << hipGetErrorString(err_t_ ## name) << std::endl; \
	} \
	float milliseconds_ ## name = 0; \
	for (int i = 0; i < NUM_RUNS; i++) \
	{ \
		CUDA_CHECK(hipMemcpy(d_C_ ## name, d_C_INI_ ## name, M * N * sizeof(float), hipMemcpyHostToDevice)); \
		hipDeviceSynchronize(); \
		hipEventRecord(start_ ## name); \
		name(d_A_ ## name, d_B_ ## name, d_C_ ## name, M, N, K); \
		hipEventRecord(end_ ## name); \
		hipEventSynchronize(end_ ## name); \
		float milliseconds_ ## i = 0; \
		hipEventElapsedTime(&milliseconds_ ## i, start_ ## name, end_ ## name); \
		milliseconds_ ## name += milliseconds_ ## i; \
	} \
	hipMemcpy(C, d_C_ ## name, M * N * sizeof(float), hipMemcpyDeviceToHost); \
	std::cout << "Time taken for GEMM (GPU, " << #name <<"): " << milliseconds_ ## name / (float)NUM_RUNS << "ms" << std::endl; \
	hipFree(d_A_ ## name); \
	hipFree(d_B_ ## name); \
	hipFree(d_C_ ## name);

__global__ void gemm_gpu_o0_kernel(float* A, float* B, float *C, int M, int N, int K) {
	// if (threadIdx.x == 0 && blockIdx.x == 0) {
	// 	for (int i = 0; i < M; i++) {
	// 		for (int j = 0; j < N; j++) {
	// 			for (int k = 0; k < K; k++) {
	// 				C[i * N + j]  += A[i * K + k]  * B[k * N + j];
	// 			}
	// 		}
	// 	}
  // }
}

void gemm_gpu_o0(float* A, float* B, float* C, int M, int N, int K)
{
	// Init block and grid size
	dim3 blockSize(1);
	dim3 gridSize(1);
	gemm_gpu_o0_kernel<<<gridSize, blockSize>>>(A, B, C, M, N, K);
}

// The scafolding for optimized GEMM implementations
__global__ void gemm_gpu_o1_kernel(float* A, float* B, float *C, int M, int N, int K) {
  int row=blockIdx.y*blockDim.y+threadIdx.y;
  int col=blockIdx.x*blockDim.x+threadIdx.x;
  if(row>=M||col>=N){
    return;
  }

  float result=0.0f;
  
  int offset_a=row*K;
  for(int i=0;i<K;i++){
      result+=A[offset_a+i]*B[i*N+col];
  }

  C[row*N+col]=result;
}

void gemm_gpu_o1(float* A, float* B, float* C, int M, int N, int K)
{
	// Init block and grid size
  dim3 blockSize(16,16);
  dim3 gridSize((N+blockSize.x-1)/blockSize.x,(M+blockSize.y-1)/blockSize.y);
	gemm_gpu_o1_kernel<<<gridSize, blockSize>>>(A, B, C, M, N, K);
}

const int BLOCK_SIZE_O2=16;

__global__ void gemm_gpu_o2_kernel(float* A, float* B, float *C, int M, int N, int K) {
  int row=blockIdx.y*blockDim.y+threadIdx.y;
  int col=blockIdx.x*blockDim.x+threadIdx.x;

  __shared__ float shared_A[BLOCK_SIZE_O2][BLOCK_SIZE_O2];
  __shared__ float shared_B[BLOCK_SIZE_O2][BLOCK_SIZE_O2];

  float result=0.0f;

  for(int offset=0;offset<K+BLOCK_SIZE_O2-1;offset+=BLOCK_SIZE_O2){
    if(row<M&&offset+threadIdx.x<K){
      shared_A[threadIdx.y][threadIdx.x]=A[row*K+offset+threadIdx.x];
    }else{
      shared_A[threadIdx.y][threadIdx.x]=0.0f;
    }
    if(col<N&&offset+threadIdx.y<K){
      shared_B[threadIdx.y][threadIdx.x]=B[(offset+threadIdx.y)*N+threadIdx.x];
    }else{
      shared_B[threadIdx.y][threadIdx.x]=0.0f;
    }
    __syncthreads();

    for(int i=0;i<BLOCK_SIZE_O2;i++) {
      result+=shared_A[threadIdx.y][i]*shared_B[i][threadIdx.x];
    }
    __syncthreads();
  }

  if(row<M&&col<N){
    C[row*N+col]=result;
  }
}

void gemm_gpu_o2(float* A, float* B, float* C, int M, int N, int K)
{
	// Init block and grid size
  dim3 blockSize(BLOCK_SIZE_O2,BLOCK_SIZE_O2);
  dim3 gridSize((N+blockSize.x-1)/blockSize.x,(M+blockSize.y-1)/blockSize.y);
	gemm_gpu_o2_kernel<<<gridSize, blockSize>>>(A, B, C, M, N, K);
}

__global__ void gemm_gpu_o3_kernel(float* A, float* B, float *C, int M, int N, int K) {
}
void gemm_gpu_o3(float* A, float* B, float* C, int M, int N, int K)
{
	// Init block and grid size
}



int main(int argc, char* argv[]) {
	if (argc < 3) {
		std::cout << "Usage: mp1 <M> <N> <K>" << std::endl;
		return 1;
	}

	int M = atoi(argv[1]);
	int N = atoi(argv[2]);
	int K = atoi(argv[3]);

	// int runs = atoi(argv[3]);
	float* A = new float[M * K]();
	float* B = new float[K * N]();
	float* C = new float[M * N]();

	fillRandom(A, M * K);
	fillRandom(B, K * N);

	/// GPU Implementation
        // Check if implementation is correct
	auto ref = Ref();
	float* refC = new float[Ref::M * Ref::N]();
 	CHECK(gemm_gpu_o0)
	CHECK(gemm_gpu_o1)
	CHECK(gemm_gpu_o2)
	CHECK(gemm_gpu_o3)

	// Actual run
 	TIME(gemm_gpu_o0)
	TIME(gemm_gpu_o1)
	TIME(gemm_gpu_o2)
	TIME(gemm_gpu_o3)

	hipHostFree(A);
	hipHostFree(B);
	hipHostFree(C);

	delete[] A;
	delete[] B;
	delete[] C;

	return 0;
}